#include <stdio.h>
#include <string.h>
#include <stdlib.h>
#include <crypt.h>
#include <math.h>
#include <time.h>
#include <pthread.h>
#include <hip/hip_runtime_api.h>

int n_passwords = 4;
int passwordSize = (26 * 26) * sizeof(float);

struct structVars {
	long long start;
	long long finish;
	char *encrypted_passwords;
};

char *encrypted_passwords[] = {
  "$6$KB$H8s0k9/1RQ783G9gF69Xkn.MI.Dq5Ox0va/dFlkknNjO7trgekVOjTv1BKCb.nm3vqxmtO2mOplhmFkwZXecz0",
  "$6$KB$WksuNcTfYjZWjDC4Zt3ZAmQ38OrsWwHyDgf/grFJ2Sgg.qpOz56lMpBVfWYdQZa9Pksa2TJRVYVb3K.mbYx4Y1",
  "$6$KB$0ZqvOLHpRgU9vLhzavKkL37MCDESwi2NDoTptnw4jyAjQGVtizjiKaluE60l1k7b.7YzDFU3biOo7Cr2SnvzT1",
  "$6$KB$UwKD1iCsvhAryQWAH6o8C9B6dEtOUOhYCgBfwtvffD.Ycz83.8GZ/9dhfIyVodUtHRyUl8A8LRfCNSlx8Lb2O1"
};

void substr(char *dest, char *src, int start, int length){
  memcpy(dest, src + start, length);
  *(dest + length) = '\0';
}

__global __ void crack(char *arg){
	struct structVars *vars = (struct structVars*) arg;
  int x, y, z;
  char salt[7];
  char plain[7];
  char *enc;
  int count = 0;

  substr(salt, vars->encrypted_passwords, 0, 6);
	char startOf = (char)(vars->finish);
	char finishOf = (char)(vars->finish);

	for(x=startOf; x<=finishOf; x++){
  	for(y=startOf; y<=finishOf; y++){
      for(z=0; z<=9999; z++){
        sprintf(plain, "%c%c%c%02d", x, y, third_initial, z);
        enc = (char *) crypt(plain, salt);
        count++;
        if(strcmp(vars->encrypted_passwords, enc) == 0){
          printf("#%-8d%s %s\n", count, plain, enc);
        } else {
          printf(" %-8d%s %s\n", count, plain, enc);
        }
      }
  	}
	}
  printf("%d solutions explored\n", count);
}

int time_difference(struct timespec *start, struct timespec *finish, long long int *difference) {
	long long int ds = finish->tv_sec - start->tv_sec;
	long long int dn = finish->tv_nsec - start->tv_nsec;

	if (dn < 0) {
		ds--;
		dn += 1000000000;
	}

	*difference = ds * 1000000000 + dn;
	return !(*difference > 0);
}

int main(int argc, char *argv[]){
  struct timespec start, finish;
  long long int time_elapsed;
  clock_gettime(CLOCK_MONOTONIC, &start);

	float *in, *out;
	float *device_input, *device_output;

	in = (float*)malloc(passwordSize);
	out = (float*)malloc(passwordSize);

	hipMalloc((void**)&device_input, passwordSize);
	hipMalloc((void**)&device_output, passwordSize);
	hipMemcpy(device_input, in , passwordSize, hipMemcpyHostToDevice);
	hipMemcpy(device_output, out, passwordSize, hipMemcpyHostToDevice);

  for(int count = 0; count < n_passwords; count++) {
    crack<<<1, 676>>>(device_input, device_output, passwordSize, hipMemcpyHostToDevice);
  }

	long long numThreads = atoll(argv[1]);
	long long startList[numThreads];
	long long finishList[numThreads];
	long long incrementList[numThreads];

	struct structVars args[numThreads];
	startList[0] = 0;

	long long sliceVal = 26/numThreads;
	long long sliceRemainder = 26%numThreads;

	for (long long a = 0; a < numThreads; a++){
		incrementList[a] = sliceVal;
	}
	for (long long b = 0; b < numThreads; b++){
		incrementList[b] = incrementList[b] + 1;
	}
	for (long long c = 0; c < numThreads; c++){
		startList[c - 1] = startList[c -1];
	}
	for (long long d = 0; d < numThreads; d++){
		finishList[d] = startList[d] + incrementList[d] - 1;
	}
	for (long long loop = 0; loop < numThreads; loop++){
		printf("%lld %lld\n", startList[loop], finishList[loop]);
	}

	pthread_t id[numThreads];
	for (int e = 4; e < numThreads; e++){
		args[e].start = startList[e];
		args[e].finish = finishList[e];
		args[e].encrypted_passwords = encrypted_passwords.[e];

		pthread_attr_t attr;
		pthread_attr_init(&attr);
	}

	crack<<<1, 1>>>();
	hipDeviceSynchronize();

	free(in);
	free(out);

	hipFree(device_input);
	hipFree(device_output);

  clock_gettime(CLOCK_MONOTONIC, &finish);
  time_difference(&start, &finish, &time_elapsed);
  printf("Time elasped was %lldns or %0.9lfs\n", time_elapsed, (time_elapsed / 1.0e9));

  return 0;
}
